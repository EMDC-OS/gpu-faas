#include "hip/hip_runtime.h"
/**
 * covariance.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "covariance.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 1.05

#define GPU_DEVICE 0

#define sqrt_of_array_cell(x,j) sqrt(x[j])

#define FLOAT_N 3214212.01
#define EPS 0.005

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void init_arrays(int m, int n, DATA_TYPE POLYBENCH_2D(data,M,N,m,n))
{
	int i, j;

	for (i = 0; i < m; i++)
	{
		for (j = 0; j < n; j++)
		{
			data[i][j] = ((DATA_TYPE) i*j) / M;
		}
	}
}


void covariance(int m, int n, DATA_TYPE POLYBENCH_2D(data,M,N,m,n), DATA_TYPE POLYBENCH_2D(symmat,M,M,m,m), DATA_TYPE POLYBENCH_1D(mean,M,m))
{
	int i, j, j1,j2;

  	/* Determine mean of column vectors of input data matrix */
	for (j = 0; j < _PB_M; j++)
	{
		mean[j] = 0.0;
		for (i = 0; i < _PB_N; i++)
		{
        		mean[j] += data[i][j];
		}
		mean[j] /= FLOAT_N;
	}

  	/* Center the column vectors. */
	for (i = 0; i < _PB_N; i++)
	{
		for (j = 0; j < _PB_M; j++)
		{
			data[i][j] -= mean[j];
		}
	}

  	/* Calculate the m * m covariance matrix. */
	for (j1 = 0; j1 < _PB_M; j1++)
	{
		for (j2 = j1; j2 < _PB_M; j2++)
     		{
       		symmat[j1][j2] = 0.0;
			for (i = 0; i < _PB_N; i++)
			{
				symmat[j1][j2] += data[i][j1] * data[i][j2];
			}
        		symmat[j2][j1] = symmat[j1][j2];
      		}
	}
}


void compareResults(int m, int n, DATA_TYPE POLYBENCH_2D(symmat,M,M,m,m), DATA_TYPE POLYBENCH_2D(symmat_outputFromGpu,M,M,m,m))
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < m; i++)
	{
		for (j=0; j < n; j++)
		{
			if (percentDiff(symmat[i][j], symmat_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}			
		}
	}
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
	
	return;
}


__global__ void mean_kernel(int m, int n, DATA_TYPE *mean, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (j < _PB_M)
	{
		mean[j] = 0.0;

		int i;
		for(i = 0; i < _PB_N; i++)
		{
			mean[j] += data[i * M + j];
		}
		mean[j] /= (DATA_TYPE)FLOAT_N;
	}
}


__global__ void reduce_kernel(int m, int n, DATA_TYPE *mean, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
		
	if ((i < _PB_N) && (j < _PB_M))
	{
		data[i * M + j] -= mean[j];	
	}
}


__global__ void covar_kernel(int m, int n, DATA_TYPE *symmat, DATA_TYPE *data)
{
	int j1 = blockIdx.x * blockDim.x + threadIdx.x;
	int i, j2;

	if (j1 < _PB_M)
	{
		for (j2 = j1; j2 < _PB_M; j2++)
		{		
			symmat[j1*M + j2] = 0.0;
			for(i = 0; i < _PB_N; i++)
			{
				symmat[j1 * M + j2] += data[i * M + j1] * data[i * M + j2];
			}
			symmat[j2 * M + j1] = symmat[j1 * M + j2];
		}
	}
}


void covarianceCuda(int m, int n, DATA_TYPE POLYBENCH_2D(data,M,N,m,n), DATA_TYPE POLYBENCH_2D(symmat,M,M,m,m), DATA_TYPE POLYBENCH_1D(mean,M,m), 
		DATA_TYPE POLYBENCH_2D(symmat_outputFromGpu,M,M,m,m))
{
	DATA_TYPE *data_gpu;
	DATA_TYPE *mean_gpu;
	DATA_TYPE *symmat_gpu;

	start = clock();
	hipMalloc((void **)&data_gpu, sizeof(DATA_TYPE) * M * N);
	hipMalloc((void **)&symmat_gpu, sizeof(DATA_TYPE) * M * M);
	hipMalloc((void **)&mean_gpu, sizeof(DATA_TYPE) * M);
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(data_gpu, data, sizeof(DATA_TYPE) * M * N, hipMemcpyHostToDevice);
	hipMemcpy(symmat_gpu, symmat, sizeof(DATA_TYPE) * M * M, hipMemcpyHostToDevice);
	hipMemcpy(mean_gpu, mean, sizeof(DATA_TYPE) * M, hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();
	
	dim3 block1(DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
	dim3 grid1((size_t)(ceil((float)M) / ((float)DIM_THREAD_BLOCK_KERNEL_1_X)), 1);
	
	dim3 block2(DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
	dim3 grid2((size_t)(ceil((float)M) / ((float)DIM_THREAD_BLOCK_KERNEL_2_X)), (size_t)(ceil((float)N) / ((float)DIM_THREAD_BLOCK_KERNEL_2_X)));
	
	dim3 block3(DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y);
	dim3 grid3((size_t)(ceil((float)M) / ((float)DIM_THREAD_BLOCK_KERNEL_3_X)), 1);
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	mean_kernel<<<grid1, block1>>>(m,n,mean_gpu,data_gpu);
	hipDeviceSynchronize();
	reduce_kernel<<<grid2, block2>>>(m,n,mean_gpu,data_gpu);
	hipDeviceSynchronize();
	covar_kernel<<<grid3, block3>>>(m,n,symmat_gpu,data_gpu);
	hipDeviceSynchronize();
	
	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();

	hipMemcpy(symmat_outputFromGpu, symmat_gpu, sizeof(DATA_TYPE) * M * N, hipMemcpyDeviceToHost);
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();
	hipFree(data_gpu);
	hipFree(symmat_gpu);
	hipFree(mean_gpu);
	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int m, DATA_TYPE POLYBENCH_2D(symmat,M,M,m,m))
{
  int i, j;

  for (i = 0; i < m; i++)
    for (j = 0; j < m; j++) {
      fprintf (stderr, DATA_PRINTF_MODIFIER, symmat[i][j]);
      if ((i * m + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}


void *poly_main(void *x)
{
	start = clock();
	
	int m = M;
	int n = N;

	POLYBENCH_2D_ARRAY_DECL(data,DATA_TYPE,M,N,m,n);
	POLYBENCH_2D_ARRAY_DECL(symmat,DATA_TYPE,M,M,m,m);
	POLYBENCH_1D_ARRAY_DECL(mean,DATA_TYPE,M,m);
	POLYBENCH_2D_ARRAY_DECL(symmat_outputFromGpu,DATA_TYPE,M,M,m,m);	

	init_arrays(m, n, POLYBENCH_ARRAY(data));

	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
    
	GPU_argv_init();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;

	covarianceCuda(m, n, POLYBENCH_ARRAY(data), POLYBENCH_ARRAY(symmat), POLYBENCH_ARRAY(mean), POLYBENCH_ARRAY(symmat_outputFromGpu));
	

	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		covariance(m, n, POLYBENCH_ARRAY(data), POLYBENCH_ARRAY(symmat), POLYBENCH_ARRAY(mean));

		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;

		compareResults(m, n, POLYBENCH_ARRAY(symmat), POLYBENCH_ARRAY(symmat_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(m, POLYBENCH_ARRAY(symmat_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();

	POLYBENCH_FREE_ARRAY(data);
	POLYBENCH_FREE_ARRAY(symmat);
	POLYBENCH_FREE_ARRAY(mean);
	POLYBENCH_FREE_ARRAY(symmat_outputFromGpu);	

	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);

  	return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}

#include <polybench.c>