#include "hip/hip_runtime.h"
/**
 * 3mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "3mm.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void init_array(int ni, int nj, int nk, int nl, int nm, DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk), DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj), 
		DATA_TYPE POLYBENCH_2D(C, NJ, NM, nj, nm), DATA_TYPE POLYBENCH_2D(D, NM, NL, nm, nl))
{
	int i, j;

	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < nk; j++)
		{
			A[i][j] = ((DATA_TYPE) i*j) / ni;
		}
	}
  
	for (i = 0; i < nk; i++)
	{
		for (j = 0; j < nj; j++)
		{
			B[i][j] = ((DATA_TYPE) i*(j+1)) / nj;
		}
	}
  
	for (i = 0; i < nj; i++)
	{
		for (j = 0; j < nm; j++)
		{
			C[i][j] = ((DATA_TYPE) i*(j+3)) / nl;
		}
	}
  
	for (i = 0; i < nm; i++)
	{
		for (j = 0; j < nl; j++)
		{
			D[i][j] = ((DATA_TYPE) i*(j+2)) / nk;
		}
	}
}


void compareResults(int ni, int nl, DATA_TYPE POLYBENCH_2D(G, NI, NL, ni, nl), DATA_TYPE POLYBENCH_2D(G_outputFromGpu, NI, NL, ni, nl))
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < ni; i++)
	{
		for (j=0; j < nl; j++)
		{
			if (percentDiff(G[i][j], G_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;				
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

	
__global__ void mm3_kernel1(int ni, int nj, int nk, int nl, int nm, DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *E)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NJ))
	{
		E[i * NJ + j] = 0;
		int k;
		for(k=0; k < _PB_NK; k++)
		{
			E[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
		}
	}
}

	
__global__ void mm3_kernel2(int ni, int nj, int nk, int nl, int nm, DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *F)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NJ) && (j < _PB_NL))
	{
		F[i * NL + j] = 0;
		int k;
		for(k=0; k < _PB_NM; k++)
		{
			F[i * NL + j] += C[i * NM + k] * D[k * NL +j];
		}
	}
}

	
__global__ void mm3_kernel3(int ni, int nj, int nk, int nl, int nm, DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NL))
	{
		G[i * NL + j] = 0;
		int k;
		for(k=0; k < _PB_NJ; k++)
		{
			G[i * NL + j] += E[i * NJ + k] * F[k * NL + j];
		}
	}
}


/* Main computational kernel on CPU */
void mm3_cpu(int ni, int nj, int nk, int nl, int nm,
		DATA_TYPE POLYBENCH_2D(E,NI,NJ,ni,nj),
		DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
		DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
		DATA_TYPE POLYBENCH_2D(F,NJ,NL,nj,nl),
		DATA_TYPE POLYBENCH_2D(C,NJ,NM,nj,nm),
		DATA_TYPE POLYBENCH_2D(D,NM,NL,nm,nl),
		DATA_TYPE POLYBENCH_2D(G,NI,NL,ni,nl))
{
	int i, j, k;

	/* E := A*B */
	for (i = 0; i < _PB_NI; i++)
	{
		for (j = 0; j < _PB_NJ; j++)
		{
			E[i][j] = 0;
			for (k = 0; k < _PB_NK; ++k)
			{
				E[i][j] += A[i][k] * B[k][j];
			}
		}
	}

	/* F := C*D */
	for (i = 0; i < _PB_NJ; i++)
	{
		for (j = 0; j < _PB_NL; j++)
		{
			F[i][j] = 0;
			for (k = 0; k < _PB_NM; ++k)
			{
				F[i][j] += C[i][k] * D[k][j];
			}
		}
	}

	/* G := E*F */
	for (i = 0; i < _PB_NI; i++)
	{
		for (j = 0; j < _PB_NL; j++)
		{
			G[i][j] = 0;
			for (k = 0; k < _PB_NJ; ++k)
			{
				G[i][j] += E[i][k] * F[k][j];
			}
		}
	}
}


void mm3Cuda(int ni, int nj, int nk, int nl, int nm,
		DATA_TYPE POLYBENCH_2D(E,NI,NJ,ni,nj),
		DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
		DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
		DATA_TYPE POLYBENCH_2D(F,NJ,NL,nj,nl),
		DATA_TYPE POLYBENCH_2D(C,NJ,NM,nj,nm),
		DATA_TYPE POLYBENCH_2D(D,NM,NL,nm,nl),
		DATA_TYPE POLYBENCH_2D(G,NI,NL,ni,nl),
		DATA_TYPE POLYBENCH_2D(G_outputFromGpu,NI,NL,ni,nl))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;
	DATA_TYPE *F_gpu;
	DATA_TYPE *G_gpu;
	
	start = clock();
	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NJ * NM);
	hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NM * NL);
	hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&F_gpu, sizeof(DATA_TYPE) * NJ * NL);
	hipMalloc((void **)&G_gpu, sizeof(DATA_TYPE) * NI * NL);
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NJ * NM, hipMemcpyHostToDevice);
	hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NM * NL, hipMemcpyHostToDevice);
	hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	hipMemcpy(F_gpu, F, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
	hipMemcpy(G_gpu, G, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NJ) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid2((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NJ/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid3((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	mm3_kernel1<<<grid1,block>>>(ni, nj, nk, nl, nm, A_gpu, B_gpu, E_gpu);
	hipDeviceSynchronize();
	mm3_kernel2<<<grid2,block>>>(ni, nj, nk, nl, nm, C_gpu, D_gpu, F_gpu);
	hipDeviceSynchronize();
	mm3_kernel3<<<grid3,block>>>(ni, nj, nk, nl, nm, E_gpu, F_gpu, G_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(G_outputFromGpu, G_gpu, sizeof(DATA_TYPE) * NI * NL, hipMemcpyDeviceToHost);
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();
	
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	hipFree(D_gpu);
	hipFree(E_gpu);
	hipFree(F_gpu);
	hipFree(G_gpu);
	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nl,
		 DATA_TYPE POLYBENCH_2D(G,NI,NL,ni,nl))
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nl; j++) {
	fprintf (stderr, DATA_PRINTF_MODIFIER, G[i][j]);
	if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}


void *poly_main(void *x)
{
	start = clock();

	int ni = NI;
	int nj = NJ;
	int nk = NK;
	int nl = NL;
	int nm = NM;

	/* Variable declaration/allocation. */
	POLYBENCH_2D_ARRAY_DECL(E, DATA_TYPE, NI, NJ, ni, nj);
	POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, NI, NK, ni, nk);
	POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, NK, NJ, nk, nj);
	POLYBENCH_2D_ARRAY_DECL(F, DATA_TYPE, NJ, NL, nj, nl);
	POLYBENCH_2D_ARRAY_DECL(C, DATA_TYPE, NJ, NM, nj, nm);
	POLYBENCH_2D_ARRAY_DECL(D, DATA_TYPE, NM, NL, nm, nl);
	POLYBENCH_2D_ARRAY_DECL(G, DATA_TYPE, NI, NL, ni, nl);
	POLYBENCH_2D_ARRAY_DECL(G_outputFromGpu, DATA_TYPE, NI, NL, ni, nl);

	init_array(ni, nj, nk, nl, nm, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D));

	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();

	GPU_argv_init();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;

	mm3Cuda(ni, nj, nk, nl, nm, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(E), 
		POLYBENCH_ARRAY(F), POLYBENCH_ARRAY(G), POLYBENCH_ARRAY(G_outputFromGpu));

	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		mm3_cpu(ni, nj, nk, nl, nm, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(D), POLYBENCH_ARRAY(E), 
			POLYBENCH_ARRAY(F), POLYBENCH_ARRAY(G));
	
		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;

		compareResults(ni, nl, POLYBENCH_ARRAY(G), POLYBENCH_ARRAY(G_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(ni, nl, POLYBENCH_ARRAY(G_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();

	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);
	POLYBENCH_FREE_ARRAY(C);
	POLYBENCH_FREE_ARRAY(D);
	POLYBENCH_FREE_ARRAY(E);
	POLYBENCH_FREE_ARRAY(F);
	POLYBENCH_FREE_ARRAY(G);
	POLYBENCH_FREE_ARRAY(G_outputFromGpu);

	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);

	return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}

#include <polybench.c>