#include "hip/hip_runtime.h"
/**
 * atax.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "atax.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0


#ifndef M_PI
#define M_PI 3.14159
#endif

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void init_array(int nx, int ny, DATA_TYPE POLYBENCH_1D(x,NX,nx), DATA_TYPE POLYBENCH_2D(A,NX,NY,nx,ny))
{
	int i, j;

	for (i = 0; i < nx; i++)
	{
		x[i] = i * M_PI;
		for (j = 0; j < ny; j++)
		{
			A[i][j] = ((DATA_TYPE) i*j) / NX;
		}
	}
}


void compareResults(int ny, DATA_TYPE POLYBENCH_1D(z,NY,ny), DATA_TYPE POLYBENCH_1D(z_outputFromGpu,NY,ny))
{
	int i, fail;
	fail = 0;

	for (i=0; i<ny; i++)
	{
		if (percentDiff(z[i], z_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}		
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void atax_kernel1(int nx, int ny, DATA_TYPE *A, DATA_TYPE *x, DATA_TYPE *tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < _PB_NX)
	{
		tmp[i] = 0;
		int j;
		for(j=0; j < _PB_NY; j++)
		{
			tmp[i] += A[i*NY+j] * x[j];
		}
	}
}

__global__ void atax_kernel2(int nx, int ny, DATA_TYPE *A, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < _PB_NY)
	{
		y[j] = 0;
		int i;
		for(i=0; i < _PB_NX; i++)
		{
			y[j] += A[i*NY+j] * tmp[i];
		}
	}
}


void atax_cpu(int nx, int ny, DATA_TYPE POLYBENCH_2D(A,NX,NY,nx,ny), DATA_TYPE POLYBENCH_1D(x,NY,ny), DATA_TYPE POLYBENCH_1D(y,NY,ny), 
		DATA_TYPE POLYBENCH_1D(tmp,NX,nx))
{
	int i,j;
	
	for (i= 0; i < _PB_NY; i++)
	{
    		y[i] = 0;
	}
  
	for (i = 0; i < _PB_NX; i++)
 	{
      		tmp[i] = 0;

      		for (j = 0; j < _PB_NY; j++)
		{
			tmp[i] = tmp[i] + A[i][j] * x[j];
		}
		
      		for (j = 0; j < _PB_NY; j++)
		{
			y[j] = y[j] + A[i][j] * tmp[i];
		}
    }
}


void ataxGpu(int nx, int ny, DATA_TYPE POLYBENCH_2D(A, NX, NY,nx,ny), DATA_TYPE POLYBENCH_1D(x,NX,nx), DATA_TYPE POLYBENCH_1D(y,NY,ny), 
		DATA_TYPE POLYBENCH_1D(tmp,NX,nx), DATA_TYPE POLYBENCH_1D(y_outputFromGpu,NY,ny))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	start = clock();
	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NX * NY);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NX);
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
	hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
	hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NX) / ((float)block.x) )), 1);
	dim3 grid2((size_t)(ceil( ((float)NY) / ((float)block.x) )), 1);
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	atax_kernel1<<< grid1, block >>>(nx, ny, A_gpu,x_gpu,tmp_gpu);
	hipDeviceSynchronize();
	atax_kernel2<<< grid2, block >>>(nx, ny, A_gpu,y_gpu,tmp_gpu);
	hipDeviceSynchronize();
	
	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost);
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();
	hipFree(A_gpu);
	hipFree(x_gpu);
	hipFree(y_gpu);
	hipFree(tmp_gpu);
	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nx, DATA_TYPE POLYBENCH_1D(y,NX,nx))
{
  int i;

  for (i = 0; i < nx; i++) {
    fprintf (stderr, DATA_PRINTF_MODIFIER, y[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
  fprintf (stderr, "\n");
}


void *poly_main(void *xxx)
{
	start = clock();

	int nx = NX;
	int ny = NY;

	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NX,NY,nx,ny);
	POLYBENCH_1D_ARRAY_DECL(x,DATA_TYPE,NY,ny);
	POLYBENCH_1D_ARRAY_DECL(y,DATA_TYPE,NY,ny);
	POLYBENCH_1D_ARRAY_DECL(y_outputFromGpu,DATA_TYPE,NY,ny);
	POLYBENCH_1D_ARRAY_DECL(tmp,DATA_TYPE,NX,nx);

	init_array(nx, ny, POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(A));

	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();

	GPU_argv_init();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;

	ataxGpu(nx, ny, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(tmp), 
		POLYBENCH_ARRAY(y_outputFromGpu));
	
	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		atax_cpu(nx, ny, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(tmp));

		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;

		compareResults(ny, POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(y_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(ny, POLYBENCH_ARRAY(y_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();

	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(x);
	POLYBENCH_FREE_ARRAY(y);
	POLYBENCH_FREE_ARRAY(y_outputFromGpu);
	POLYBENCH_FREE_ARRAY(tmp);

	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);

  	return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}

#include <polybench.c>