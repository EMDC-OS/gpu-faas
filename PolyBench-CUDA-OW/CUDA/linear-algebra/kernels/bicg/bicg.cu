#include "hip/hip_runtime.h"
/**
 * bicg.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "bicg.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//Error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

#ifndef M_PI
#define M_PI 3.14159
#endif

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void init_array(int nx, int ny, DATA_TYPE POLYBENCH_2D(A,NX,NY,nx,ny), DATA_TYPE POLYBENCH_1D(p,NY,ny), DATA_TYPE POLYBENCH_1D(r,NX,nx))
{
	int i, j;
	
	for (i = 0; i < ny; i++)
	{
    		p[i] = i * M_PI;
	}

	for (i = 0; i < nx; i++)
	{
    		r[i] = i * M_PI;

    		for (j = 0; j < ny; j++)
		{
      			A[i][j] = ((DATA_TYPE) i*j) / NX;
		}
 	}
}


void compareResults(int nx, int ny, DATA_TYPE POLYBENCH_1D(s,NY,ny), DATA_TYPE POLYBENCH_1D(s_outputFromGpu,NY,ny), 
		DATA_TYPE POLYBENCH_1D(q,NX,nx), DATA_TYPE POLYBENCH_1D(q_outputFromGpu,NX,nx))
{
	int i,fail;
	fail = 0;

	// Compare s with s_cuda
	for (i=0; i<nx; i++)
	{
		if (percentDiff(q[i], q_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}
	}

	for (i=0; i<ny; i++)
	{
		if (percentDiff(s[i], s_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}		
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


//Distributed (split) from initial loop and permuted into reverse order to allow parallelism...
__global__ void bicg_kernel1(int nx, int ny, DATA_TYPE *A, DATA_TYPE *r, DATA_TYPE *s)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < _PB_NY)
	{
		s[j] = 0.0f;

		int i;
		for(i = 0; i < _PB_NX; i++)
		{
			s[j] += r[i] * A[i * NY + j];
		}
	}	
}


//Distributed (split) from initial loop to allow parallelism
__global__ void bicg_kernel2(int nx, int ny, DATA_TYPE *A, DATA_TYPE *p, DATA_TYPE *q)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < _PB_NX)
	{
		q[i] = 0.0f;

		int j;
		for(j=0; j < _PB_NY; j++)
		{
			q[i] += A[i * NY + j] * p[j];
		}
	}
}


void bicg_cpu(int nx, int ny, DATA_TYPE POLYBENCH_2D(A,NX,NY,nx,ny), DATA_TYPE POLYBENCH_1D(r,NX,nx), DATA_TYPE POLYBENCH_1D(s,NY,ny), 
		DATA_TYPE POLYBENCH_1D(p,NY,ny), DATA_TYPE POLYBENCH_1D(q,NX,nx))
{
	int i,j;
	
  	for (i = 0; i < _PB_NY; i++)
	{
		s[i] = 0.0;
	}

	for (i = 0; i < _PB_NX; i++)
	{
		q[i] = 0.0;
		for (j = 0; j < _PB_NY; j++)
	  	{
	    		s[j] = s[j] + r[i] * A[i][j];
	    		q[i] = q[i] + A[i][j] * p[j];
	  	}
	}
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nx, int ny,
		 DATA_TYPE POLYBENCH_1D(s,NY,ny),
		 DATA_TYPE POLYBENCH_1D(q,NX,nx))

{
  int i;

  for (i = 0; i < ny; i++) {
    fprintf (stderr, DATA_PRINTF_MODIFIER, s[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
  for (i = 0; i < nx; i++) {
    fprintf (stderr, DATA_PRINTF_MODIFIER, q[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
  fprintf (stderr, "\n");
}


void bicgCuda(int nx, int ny, DATA_TYPE POLYBENCH_2D(A,NX,NY,nx,ny), DATA_TYPE POLYBENCH_1D(r,NX,nx), DATA_TYPE POLYBENCH_1D(s,NY,ny), 
	DATA_TYPE POLYBENCH_1D(p,NY,ny), DATA_TYPE POLYBENCH_1D(q,NX,nx), DATA_TYPE POLYBENCH_1D(s_outputFromGpu,NY,ny), 
	DATA_TYPE POLYBENCH_1D(q_outputFromGpu,NX,nx))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *q_gpu;
	DATA_TYPE *p_gpu;
	DATA_TYPE *r_gpu;
	DATA_TYPE *s_gpu;

	start = clock();
	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NX * NY);
	hipMalloc((void **)&r_gpu, sizeof(DATA_TYPE) * NX);
	hipMalloc((void **)&s_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&p_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&q_gpu, sizeof(DATA_TYPE) * NX);
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
	hipMemcpy(r_gpu, r, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice);
	hipMemcpy(s_gpu, s, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
	hipMemcpy(p_gpu, p, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
	hipMemcpy(q_gpu, q, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NY) / ((float)block.x) )), 1);
	dim3 grid2((size_t)(ceil( ((float)NX) / ((float)block.x) )), 1);
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	bicg_kernel1<<< grid1, block >>>(nx, ny, A_gpu, r_gpu, s_gpu);
	hipDeviceSynchronize();
	bicg_kernel2<<< grid2, block >>>(nx, ny, A_gpu, p_gpu, q_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	
	hipMemcpy(s_outputFromGpu, s_gpu, sizeof(DATA_TYPE) * NY, hipMemcpyDeviceToHost);
	hipMemcpy(q_outputFromGpu, q_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost);

	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();
	hipFree(A_gpu);
	hipFree(r_gpu);
	hipFree(s_gpu);
	hipFree(p_gpu);
	hipFree(q_gpu);
	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


void *poly_main(void *x)
{
	start = clock();

	int nx = NX;
	int ny = NY;

	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NX,NY,nx,ny);
	POLYBENCH_1D_ARRAY_DECL(s,DATA_TYPE,NY,ny);
	POLYBENCH_1D_ARRAY_DECL(q,DATA_TYPE,NX,nx);
	POLYBENCH_1D_ARRAY_DECL(p,DATA_TYPE,NY,ny);
	POLYBENCH_1D_ARRAY_DECL(r,DATA_TYPE,NX,nx);
	POLYBENCH_1D_ARRAY_DECL(s_outputFromGpu,DATA_TYPE,NY,ny);
	POLYBENCH_1D_ARRAY_DECL(q_outputFromGpu,DATA_TYPE,NX,nx);

	init_array(nx, ny, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(p), POLYBENCH_ARRAY(r));

	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();

	GPU_argv_init();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;

	bicgCuda(nx, ny, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(r), POLYBENCH_ARRAY(s), POLYBENCH_ARRAY(p), POLYBENCH_ARRAY(q), 
		POLYBENCH_ARRAY(s_outputFromGpu), POLYBENCH_ARRAY(q_outputFromGpu));

	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		bicg_cpu(nx, ny, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(r), POLYBENCH_ARRAY(s), POLYBENCH_ARRAY(p), POLYBENCH_ARRAY(q));

		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;
	
		compareResults(nx, ny, POLYBENCH_ARRAY(s), POLYBENCH_ARRAY(s_outputFromGpu), POLYBENCH_ARRAY(q), 
			POLYBENCH_ARRAY(q_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(nx, ny, POLYBENCH_ARRAY(s_outputFromGpu), POLYBENCH_ARRAY(q_outputFromGpu)));
	
	#endif //RUN_ON_CPU

	start = clock();

	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(r);
	POLYBENCH_FREE_ARRAY(s);
	POLYBENCH_FREE_ARRAY(p);
	POLYBENCH_FREE_ARRAY(q);
	POLYBENCH_FREE_ARRAY(s_outputFromGpu);
	POLYBENCH_FREE_ARRAY(q_outputFromGpu);

	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);

  	return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}

#include <polybench.c>