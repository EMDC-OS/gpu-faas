#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "gemm.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void gemm(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk), 
	 DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj), DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj))
{
	int i,j,k;
	
	for (i = 0; i < _PB_NI; i++)
	{
    		for (j = 0; j < _PB_NJ; j++)
    		{
			C[i][j] *= beta;
	
			for (k = 0; k < _PB_NK; ++k)
			{
	  			C[i][j] += alpha * A[i][k] * B[k][j];
			}
      		}
	}
}


void init(int ni, int nj, int nk, DATA_TYPE* alpha, DATA_TYPE* beta, DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk), 
	DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj), DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj))
{
	int i, j;

	*alpha = 32412;
	*beta = 2123;

  	for (i = 0; i < ni; i++)
	{
    		for (j = 0; j < nk; j++)
		{
      			A[i][j] = ((DATA_TYPE) i*j) / NI;
		}
	}

  	for (i = 0; i < nk; i++)
	{
    		for (j = 0; j < nj; j++)
		{
      			B[i][j] = ((DATA_TYPE) i*j) / NI;
		}
	}

  	for (i = 0; i < ni; i++)
	{
    		for (j = 0; j < nj; j++)
		{
      			C[i][j] = ((DATA_TYPE) i*j) / NI;
		}
	}
}


void compareResults(int ni, int nj, DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj), DATA_TYPE POLYBENCH_2D(C_outputFromGpu,NI,NJ,ni,nj))
{
	int i, j, fail;
	fail = 0;
	
	// Compare CPU and GPU outputs
	for (i=0; i < ni; i++) 
	{
		for (j=0; j < nj; j++) 
		{
			if (percentDiff(C[i][j], C_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gemm_kernel(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NI) && (j < _PB_NJ))
	{	
		c[i * NJ + j] *= beta;
		int k;
		for(k=0; k < _PB_NK; k++)
		{
			c[i * NJ + j] += alpha * a[i * NK + k] * b[k * NJ +j];
		}
	}
}


void gemmCuda(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk), 
	DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj), DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj), DATA_TYPE POLYBENCH_2D(C_outputFromGpu,NI,NJ,ni,nj))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	start = clock();
	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(ceil( ((float)NI)/ ((float)block.x) )),(size_t)(ceil( ((float)NJ)/ ((float)block.y) )));
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	gemm_kernel<<< grid, block >>>(ni, nj, nk, alpha, beta, A_gpu, B_gpu, C_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);    
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nj,
		 DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj))
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
	fprintf (stderr, DATA_PRINTF_MODIFIER, C[i][j]);
	if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}


void *poly_main(void *x)
{
	start = clock();
	/* Retrieve problem size. */
	int ni = NI;
	int nj = NJ;
	int nk = NK;

	/* Variable declaration/allocation. */
	DATA_TYPE alpha;
	DATA_TYPE beta;
	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NK,ni,nk);
	POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NK,NJ,nk,nj);
	POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(C_outputFromGpu,DATA_TYPE,NI,NJ,ni,nj);

	init(ni, nj, nk, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	
	GPU_argv_init();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;
	
	gemmCuda(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));


	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		gemm(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));
		
		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
  		polybench_stop_instruments;
	 	polybench_print_instruments;
	
		compareResults(ni, nj, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(ni, nj, POLYBENCH_ARRAY(C_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();

	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);  
	POLYBENCH_FREE_ARRAY(C);  
	POLYBENCH_FREE_ARRAY(C_outputFromGpu);

	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear); 

    return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}

#include <polybench.c>