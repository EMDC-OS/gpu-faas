#include "hip/hip_runtime.h"
/**
 * gemver.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "gemver.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void gemver(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, N, N, n, n), DATA_TYPE POLYBENCH_1D(u1, N, n), DATA_TYPE POLYBENCH_1D(v1, N, n), 
	DATA_TYPE POLYBENCH_1D(u2, N, n), DATA_TYPE POLYBENCH_1D(v2, N, n), DATA_TYPE POLYBENCH_1D(w, N, n), DATA_TYPE POLYBENCH_1D(x, N, n), DATA_TYPE POLYBENCH_1D(y, N, n), 
	DATA_TYPE POLYBENCH_1D(z, N, n))
{
	int i,j;
	
  	for (i = 0; i < _PB_N; i++)
	{
    		for (j = 0; j < _PB_N; j++)
		{
      			A[i][j] = A[i][j] + u1[i] * v1[j] + u2[i] * v2[j];
		}
	}

  	for (i = 0; i < _PB_N; i++)
	{
    		for (j = 0; j < _PB_N; j++)
		{
      			x[i] = x[i] + beta * A[j][i] * y[j];
		}
	}

  	for (i = 0; i < _PB_N; i++)
	{
    		x[i] = x[i] + z[i];
	}

  	for (i = 0; i < _PB_N; i++)
	{
    		for (j = 0; j < _PB_N; j++)
		{
      			w[i] = w[i] +  alpha * A[i][j] * x[j];
		}
	}
}


void init(int n, DATA_TYPE *alpha,
	DATA_TYPE *beta,
	DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
	DATA_TYPE POLYBENCH_1D(u1,N,n),
	DATA_TYPE POLYBENCH_1D(v1,N,n),
	DATA_TYPE POLYBENCH_1D(u2,N,n),
	DATA_TYPE POLYBENCH_1D(v2,N,n),
	DATA_TYPE POLYBENCH_1D(w,N,n),
	DATA_TYPE POLYBENCH_1D(x,N,n),
	DATA_TYPE POLYBENCH_1D(y,N,n),
	DATA_TYPE POLYBENCH_1D(z,N,n))
{
	int i, j;

	*alpha = 43532;
	*beta = 12313;

  	for (i = 0; i < N; i++)
	{
	    	u1[i] = i;
	    	u2[i] = (i+1)/N/2.0;
	    	v1[i] = (i+1)/N/4.0;
	    	v2[i] = (i+1)/N/6.0;
	    	y[i] = (i+1)/N/8.0;
	    	z[i] = (i+1)/N/9.0;
	    	x[i] = 0.0;
	    	w[i] = 0.0;

    		for (j = 0; j < N; j++)
		{
			A[i][j] = ((DATA_TYPE) i*j) / N;
		}
	}
}


void compareResults(int n, DATA_TYPE POLYBENCH_1D(w1, N, n), DATA_TYPE POLYBENCH_1D(w2, N, n))
{
	int i, fail;
	fail = 0;
	
	for (i=0; i < N; i++) 
	{
		if (percentDiff(w1[i], w2[i]) > PERCENT_DIFF_ERROR_THRESHOLD) 
		{
			fail++;
		}
	}
		
	// Print results
	printf("Number of misses: %d\n", fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gemver_kernel1(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *v1, DATA_TYPE *v2, DATA_TYPE *u1, DATA_TYPE *u2)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_N) && (j < _PB_N))
	{
		a[i * N + j] += u1[i] * v1[j] + u2[i] * v2[j];
	}
}


__global__ void gemver_kernel2(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *x, DATA_TYPE *y, DATA_TYPE *z)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < _PB_N)
	{
		int j;
		for(j = 0; j < _PB_N; j++) 
		{
			x[i] += beta * a[j * N + i] * y[j];
		}
		x[i] += z[i];
	}
}


__global__ void gemver_kernel3(int n, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE *a, DATA_TYPE *x, DATA_TYPE *w)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((i >= 0) && (i < _PB_N))
	{
		int j;
		for(j = 0; j < _PB_N; j++)
		{ 
			w[i] += alpha * a[i*N + j] * x[j];
		}
	}
}


void gemverCuda(int n, DATA_TYPE alpha, DATA_TYPE beta,
		DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		DATA_TYPE POLYBENCH_1D(u1,N,n),
		DATA_TYPE POLYBENCH_1D(v1,N,n),
		DATA_TYPE POLYBENCH_1D(u2,N,n),
		DATA_TYPE POLYBENCH_1D(v2,N,n),
		DATA_TYPE POLYBENCH_1D(w,N,n),
		DATA_TYPE POLYBENCH_1D(w_outputFromGpu,N,n),
		DATA_TYPE POLYBENCH_1D(x,N,n),
		DATA_TYPE POLYBENCH_1D(y,N,n),
		DATA_TYPE POLYBENCH_1D(z,N,n))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *z_gpu;
	DATA_TYPE *v1_gpu;
	DATA_TYPE *v2_gpu;
	DATA_TYPE *u1_gpu;
	DATA_TYPE *u2_gpu;
	DATA_TYPE *w_gpu;

	start = clock();
	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&z_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&w_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&v1_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&v2_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&u1_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&u2_gpu, sizeof(DATA_TYPE) * N);
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(z_gpu, z, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(w_gpu, w, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(v1_gpu, v1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(v2_gpu, v2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(u1_gpu, u1, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(u2_gpu, u2, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();

	dim3 block1(DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
	dim3 grid1((size_t)(ceil((float)N) / ((float)DIM_THREAD_BLOCK_KERNEL_1_X)), (size_t)(ceil((float)N) / ((float)DIM_THREAD_BLOCK_KERNEL_1_Y)));

	dim3 block2(DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
	dim3 grid2((size_t)(ceil((float)N) / ((float)DIM_THREAD_BLOCK_KERNEL_2_X)), 1);
	
	dim3 block3(DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y);
	dim3 grid3((size_t)(ceil((float)N) / ((float)DIM_THREAD_BLOCK_KERNEL_3_X)), 1);
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	gemver_kernel1<<< grid1, block1 >>>(n, alpha, beta, A_gpu,v1_gpu,v2_gpu, u1_gpu, u2_gpu);
	hipDeviceSynchronize();
	gemver_kernel2<<< grid2, block2 >>>(n, alpha, beta, A_gpu,x_gpu,y_gpu, z_gpu);
	hipDeviceSynchronize();
	gemver_kernel3<<< grid3, block3 >>>(n, alpha, beta, A_gpu,x_gpu,w_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(w_outputFromGpu, w_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();
	hipFree(A_gpu);
	hipFree(x_gpu);
	hipFree(y_gpu);
	hipFree(z_gpu);
	hipFree(w_gpu);
	hipFree(v1_gpu);
	hipFree(v2_gpu);
	hipFree(u1_gpu);
	hipFree(u2_gpu);
	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(w,N,n))
{
  int i;

  for (i = 0; i < n; i++) {
    fprintf (stderr, DATA_PRINTF_MODIFIER, w[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
}
	

void *poly_main(void *xxx)
{
	start = clock();

	/* Retrieve problem size. */
	int n = N;

	/* Variable declaration/allocation. */
	DATA_TYPE alpha;
	DATA_TYPE beta;

	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,N,N,n,n);
	POLYBENCH_1D_ARRAY_DECL(u1,DATA_TYPE,N,n);
  	POLYBENCH_1D_ARRAY_DECL(v1,DATA_TYPE,N,n);
  	POLYBENCH_1D_ARRAY_DECL(u2,DATA_TYPE,N,n);
  	POLYBENCH_1D_ARRAY_DECL(v2,DATA_TYPE,N,n);
  	POLYBENCH_1D_ARRAY_DECL(w,DATA_TYPE,N,n);
  	POLYBENCH_1D_ARRAY_DECL(w_outputFromGpu,DATA_TYPE,N,n);
  	POLYBENCH_1D_ARRAY_DECL(x,DATA_TYPE,N,n);
  	POLYBENCH_1D_ARRAY_DECL(y,DATA_TYPE,N,n);
  	POLYBENCH_1D_ARRAY_DECL(z,DATA_TYPE,N,n);
  	
	
	init(n, &alpha, &beta,
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(u1),
	      POLYBENCH_ARRAY(v1),
	      POLYBENCH_ARRAY(u2),
	      POLYBENCH_ARRAY(v2),
	      POLYBENCH_ARRAY(w),
	      POLYBENCH_ARRAY(x),
	      POLYBENCH_ARRAY(y),
	      POLYBENCH_ARRAY(z));
	
	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();

	GPU_argv_init();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;

	gemverCuda(n, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(u1), POLYBENCH_ARRAY(v1), POLYBENCH_ARRAY(u2), POLYBENCH_ARRAY(v2), 
		POLYBENCH_ARRAY(w), POLYBENCH_ARRAY(w_outputFromGpu), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(z));

	#ifdef RUN_ON_CPU

	 	/* Start timer. */
	  	polybench_start_instruments;
	
		gemver(n, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(u1), POLYBENCH_ARRAY(v1), POLYBENCH_ARRAY(u2), POLYBENCH_ARRAY(v2), 
			POLYBENCH_ARRAY(w), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(z));


		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
  		polybench_stop_instruments;
 		polybench_print_instruments;
		
		compareResults(n, POLYBENCH_ARRAY(w), POLYBENCH_ARRAY(w_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(w_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();

	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(w);  
	POLYBENCH_FREE_ARRAY(w_outputFromGpu);  
	POLYBENCH_FREE_ARRAY(x);  
	POLYBENCH_FREE_ARRAY(y);
	POLYBENCH_FREE_ARRAY(z);
	POLYBENCH_FREE_ARRAY(u1);
	POLYBENCH_FREE_ARRAY(u2);
	POLYBENCH_FREE_ARRAY(v1);
	POLYBENCH_FREE_ARRAY(v2);

	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);

 	return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}

#include <polybench.c>