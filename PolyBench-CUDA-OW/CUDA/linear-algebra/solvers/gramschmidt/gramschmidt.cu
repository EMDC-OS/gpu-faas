#include "hip/hip_runtime.h"
/**
 * gramschmidt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "gramschmidt.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void gramschmidt(int ni, int nj, DATA_TYPE POLYBENCH_2D(A,NI,NJ,ni,nj), DATA_TYPE POLYBENCH_2D(R,NJ,NJ,nj,nj), DATA_TYPE POLYBENCH_2D(Q,NI,NJ,ni,nj))
{
	int i,j,k;
	DATA_TYPE nrm;
	for (k = 0; k < _PB_NJ; k++)
	{
		nrm = 0;
		for (i = 0; i < _PB_NI; i++)
		{
			nrm += A[i][k] * A[i][k];
		}
		
		R[k][k] = sqrt(nrm);
		for (i = 0; i < _PB_NI; i++)
		{
			Q[i][k] = A[i][k] / R[k][k];
		}
		
		for (j = k + 1; j < _PB_NJ; j++)
		{
			R[k][j] = 0;
			for (i = 0; i < _PB_NI; i++)
			{
				R[k][j] += Q[i][k] * A[i][j];
			}
			for (i = 0; i < _PB_NI; i++)
			{
				A[i][j] = A[i][j] - Q[i][k] * R[k][j];
			}
		}
	}
}

/* Array initialization. */
void init_array(int ni, int nj,
		DATA_TYPE POLYBENCH_2D(A,NI,NJ,ni,nj),
		DATA_TYPE POLYBENCH_2D(R,NJ,NJ,nj,nj),
		DATA_TYPE POLYBENCH_2D(Q,NI,NJ,ni,nj))
{
	int i, j;

	for (i = 0; i < ni; i++)
	{
		for (j = 0; j < nj; j++) 
		{
			A[i][j] = ((DATA_TYPE) i*j) / ni;
			Q[i][j] = ((DATA_TYPE) i*(j+1)) / nj;
		}
	}

	for (i = 0; i < nj; i++)
	{
		for (j = 0; j < nj; j++)
		{
			R[i][j] = ((DATA_TYPE) i*(j+2)) / nj;
		}
	}
}

void compareResults(int ni, int nj, DATA_TYPE POLYBENCH_2D(A,NI,NJ,ni,nj), DATA_TYPE POLYBENCH_2D(A_outputFromGpu,NI,NJ,ni,nj))
{
	int i, j, fail;
	fail = 0;

	for (i=0; i < ni; i++) 
	{
		for (j=0; j < nj; j++) 
		{
			if (percentDiff(A[i][j], A_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{				
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );	
	return;
}


__global__ void gramschmidt_kernel1(int ni, int nj, DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid==0)
	{
		DATA_TYPE nrm = 0.0;
		int i;
		for (i = 0; i < _PB_NI; i++)
		{
			nrm += a[i * NJ + k] * a[i * NJ + k];
		}
      		r[k * NJ + k] = sqrt(nrm);
	}
}


__global__ void gramschmidt_kernel2(int ni, int nj, DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < _PB_NI)
	{	
		q[i * NJ + k] = a[i * NJ + k] / r[k * NJ + k];
	}
}


__global__ void gramschmidt_kernel3(int ni, int nj, DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if ((j > k) && (j < _PB_NJ))
	{
		r[k*NJ + j] = 0.0;

		int i;
		for (i = 0; i < _PB_NI; i++)
		{
			r[k*NJ + j] += q[i*NJ + k] * a[i*NJ + j];
		}
		
		for (i = 0; i < _PB_NI; i++)
		{
			a[i*NJ + j] -= q[i*NJ + k] * r[k*NJ + j];
		}
	}
}


void gramschmidtCuda(int ni, int nj, DATA_TYPE POLYBENCH_2D(A,NI,NJ,ni,nj), DATA_TYPE POLYBENCH_2D(R,NJ,NJ,nj,nj), DATA_TYPE POLYBENCH_2D(Q,NI,NJ,ni,nj), DATA_TYPE POLYBENCH_2D(A_outputFromGpu,NI,NJ,ni,nj))
{
	start = clock();
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 gridKernel1(1, 1);
	dim3 gridKernel2((size_t)ceil(((float)NJ) / ((float)DIM_THREAD_BLOCK_X)), 1);
	dim3 gridKernel3((size_t)ceil(((float)NJ) / ((float)DIM_THREAD_BLOCK_X)), 1);
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;
	
	DATA_TYPE *A_gpu;
	DATA_TYPE *R_gpu;
	DATA_TYPE *Q_gpu;

	start = clock();
	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&R_gpu, sizeof(DATA_TYPE) * NJ * NJ);
	hipMalloc((void **)&Q_gpu, sizeof(DATA_TYPE) * NI * NJ);
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();
	
	/* Start timer. */
  	//polybench_start_instruments;
	int k;
	for (k = 0; k < _PB_NJ; k++)
	{
		gramschmidt_kernel1<<<gridKernel1,block>>>(ni, nj, A_gpu, R_gpu, Q_gpu, k);
		hipDeviceSynchronize();
		gramschmidt_kernel2<<<gridKernel2,block>>>(ni, nj, A_gpu, R_gpu, Q_gpu, k);
		hipDeviceSynchronize();
		gramschmidt_kernel3<<<gridKernel3,block>>>(ni, nj, A_gpu, R_gpu, Q_gpu, k);
		hipDeviceSynchronize();
	}
	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();
	hipMemcpy(A_outputFromGpu, A_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);   
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock(); 

	hipFree(A_gpu);
	hipFree(R_gpu);
	hipFree(Q_gpu);

	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nj, DATA_TYPE POLYBENCH_2D(A,NI,NJ,ni,nj))
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
	fprintf (stderr, DATA_PRINTF_MODIFIER, A[i][j]);
	if (i % 20 == 0) fprintf (stderr, "\n");
    }

  fprintf (stderr, "\n");
}


void *poly_main(void *x)
{
	start = clock();

	/* Retrieve problem size. */
	int ni = NI;
	int nj = NJ;

	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NJ,ni,nj);
  	POLYBENCH_2D_ARRAY_DECL(A_outputFromGpu,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(R,DATA_TYPE,NJ,NJ,nj,nj);
	POLYBENCH_2D_ARRAY_DECL(Q,DATA_TYPE,NI,NJ,ni,nj);
	
	init_array(ni, nj, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(R), POLYBENCH_ARRAY(Q));

	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	
	GPU_argv_init();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;

	gramschmidtCuda(ni, nj, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(R), POLYBENCH_ARRAY(Q), POLYBENCH_ARRAY(A_outputFromGpu));

	#ifdef RUN_ON_CPU
	
		/* Start timer. */
	  	polybench_start_instruments;

		gramschmidt(ni, nj, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(R), POLYBENCH_ARRAY(Q));

		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;
	
		compareResults(ni, nj, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(A_outputFromGpu));
	
	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(ni, nj, POLYBENCH_ARRAY(A_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();

	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(A_outputFromGpu);
	POLYBENCH_FREE_ARRAY(R);
	POLYBENCH_FREE_ARRAY(Q);  

	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);

    return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}
#include <polybench.c>