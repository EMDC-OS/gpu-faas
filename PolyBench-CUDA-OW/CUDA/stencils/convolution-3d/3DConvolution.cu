#include "hip/hip_runtime.h"
/**
 * 3DConvolution.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "3DConvolution.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void conv3D(int ni, int nj, int nk, DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk), DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk))
{
	int i, j, k;
	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +2;  c21 = +5;  c31 = -8;
	c12 = -3;  c22 = +6;  c32 = -9;
	c13 = +4;  c23 = +7;  c33 = +10;

	for (i = 1; i < _PB_NI - 1; ++i) // 0
	{
		for (j = 1; j < _PB_NJ - 1; ++j) // 1
		{
			for (k = 1; k < _PB_NK -1; ++k) // 2
			{
				B[i][j][k] = c11 * A[(i - 1)][(j - 1)][(k - 1)]  +  c13 * A[(i + 1)][(j - 1)][(k - 1)]
					     +   c21 * A[(i - 1)][(j - 1)][(k - 1)]  +  c23 * A[(i + 1)][(j - 1)][(k - 1)]
					     +   c31 * A[(i - 1)][(j - 1)][(k - 1)]  +  c33 * A[(i + 1)][(j - 1)][(k - 1)]
					     +   c12 * A[(i + 0)][(j - 1)][(k + 0)]  +  c22 * A[(i + 0)][(j + 0)][(k + 0)]   
					     +   c32 * A[(i + 0)][(j + 1)][(k + 0)]  +  c11 * A[(i - 1)][(j - 1)][(k + 1)]  
					     +   c13 * A[(i + 1)][(j - 1)][(k + 1)]  +  c21 * A[(i - 1)][(j + 0)][(k + 1)]  
					     +   c23 * A[(i + 1)][(j + 0)][(k + 1)]  +  c31 * A[(i - 1)][(j + 1)][(k + 1)]  
					     +   c33 * A[(i + 1)][(j + 1)][(k + 1)];
			}
		}
	}
}


void init(int ni, int nj, int nk, DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk))
{
	int i, j, k;

	for (i = 0; i < ni; ++i)
    	{
		for (j = 0; j < nj; ++j)
		{
			for (k = 0; k < nk; ++k)
			{
				A[i][j][k] = i % 12 + 2 * (j % 7) + 3 * (k % 13);
			}
		}
	}
}


void compareResults(int ni, int nj, int nk, DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk), DATA_TYPE POLYBENCH_3D(B_outputFromGpu, NI, NJ, NK, ni, nj, nk))
{
	int i, j, k, fail;
	fail = 0;
	
	// Compare result from cpu and gpu
	for (i = 1; i < ni - 1; ++i) // 0
	{
		for (j = 1; j < nj - 1; ++j) // 1
		{
			for (k = 1; k < nk - 1; ++k) // 2
			{
				if (percentDiff(B[i][j][k], B_outputFromGpu[i][j][k]) > PERCENT_DIFF_ERROR_THRESHOLD)
				{
					fail++;
				}
			}	
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void convolution3D_kernel(int ni, int nj, int nk, DATA_TYPE* A, DATA_TYPE* B, int i)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	DATA_TYPE c11, c12, c13, c21, c22, c23, c31, c32, c33;

	c11 = +2;  c21 = +5;  c31 = -8;
	c12 = -3;  c22 = +6;  c32 = -9;
	c13 = +4;  c23 = +7;  c33 = +10;


	if ((i < (_PB_NI-1)) && (j < (_PB_NJ-1)) &&  (k < (_PB_NK-1)) && (i > 0) && (j > 0) && (k > 0))
	{
		B[i*(NK * NJ) + j*NK + k] = c11 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]  +  c13 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]
					     +   c21 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]  +  c23 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]
					     +   c31 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]  +  c33 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k - 1)]
					     +   c12 * A[(i + 0)*(NK * NJ) + (j - 1)*NK + (k + 0)]  +  c22 * A[(i + 0)*(NK * NJ) + (j + 0)*NK + (k + 0)]   
					     +   c32 * A[(i + 0)*(NK * NJ) + (j + 1)*NK + (k + 0)]  +  c11 * A[(i - 1)*(NK * NJ) + (j - 1)*NK + (k + 1)]  
					     +   c13 * A[(i + 1)*(NK * NJ) + (j - 1)*NK + (k + 1)]  +  c21 * A[(i - 1)*(NK * NJ) + (j + 0)*NK + (k + 1)]  
					     +   c23 * A[(i + 1)*(NK * NJ) + (j + 0)*NK + (k + 1)]  +  c31 * A[(i - 1)*(NK * NJ) + (j + 1)*NK + (k + 1)]  
					     +   c33 * A[(i + 1)*(NK * NJ) + (j + 1)*NK + (k + 1)];
	}
}


void convolution3DCuda(int ni, int nj, int nk, DATA_TYPE POLYBENCH_3D(A, NI, NJ, NK, ni, nj, nk), DATA_TYPE POLYBENCH_3D(B, NI, NJ, NK, ni, nj, nk), DATA_TYPE POLYBENCH_3D(B_outputFromGpu, NI, NJ, NK, ni, nj, nk))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;

	start = clock();
	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK);
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NJ * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NI * NJ * NK, hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(ceil( ((float)NK) / ((float)block.x) )), (size_t)(ceil( ((float)NJ) / ((float)block.y) )));
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	int i;
	for (i = 1; i < _PB_NI - 1; ++i) // 0
	{
		convolution3D_kernel<<< grid, block >>>(ni, nj, nk, A_gpu, B_gpu, i);
	}

	hipDeviceSynchronize();
	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(B_outputFromGpu, B_gpu, sizeof(DATA_TYPE) * NI * NJ * NK, hipMemcpyDeviceToHost);
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();
	
	hipFree(A_gpu);
	hipFree(B_gpu);

	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nj, int nk,
		 DATA_TYPE POLYBENCH_3D(B,NI,NJ,NK,ni,nj,nk))
{
  int i, j, k;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) 
	for (k = 0; k < nk; k++)
	{
	fprintf (stderr, DATA_PRINTF_MODIFIER, B[i][j][k]);
	if ((i * (nj*nk) + j*nk + k) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}


void *poly_main(void *x)
{
	start = clock();

	int ni = NI;
	int nj = NJ;
	int nk = NK;

	POLYBENCH_3D_ARRAY_DECL(A,DATA_TYPE,NI,NJ,NK,ni,nj,nk);
	POLYBENCH_3D_ARRAY_DECL(B,DATA_TYPE,NI,NJ,NK,ni,nj,nk);
	POLYBENCH_3D_ARRAY_DECL(B_outputFromGpu,DATA_TYPE,NI,NJ,NK,ni,nj,nk);

	init(ni, nj, nk, POLYBENCH_ARRAY(A));

	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	
	GPU_argv_init();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;

	convolution3DCuda(ni, nj, nk, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_outputFromGpu));

	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		conv3D(ni, nj, nk, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;
	
		compareResults(ni, nj, nk, POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(B_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(NI, NJ, NK, POLYBENCH_ARRAY(B_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();

	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);
	POLYBENCH_FREE_ARRAY(B_outputFromGpu);

	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);

    	return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}
#include <polybench.c>