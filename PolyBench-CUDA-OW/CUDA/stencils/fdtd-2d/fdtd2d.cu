#include "hip/hip_runtime.h"
/**
 * fdtd2d.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "fdtd2d.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 10.05

#define GPU_DEVICE 0

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void init_arrays(int tmax, int nx, int ny, DATA_TYPE POLYBENCH_1D(_fict_, TMAX, TMAX), DATA_TYPE POLYBENCH_2D(ex,NX,NY,nx,ny), 
		DATA_TYPE POLYBENCH_2D(ey,NX,NY,nx,ny), DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny))
{
	int i, j;

  	for (i = 0; i < tmax; i++)
	{
		_fict_[i] = (DATA_TYPE) i;
	}
	
	for (i = 0; i < nx; i++)
	{
		for (j = 0; j < ny; j++)
		{
			ex[i][j] = ((DATA_TYPE) i*(j+1) + 1) / NX;
			ey[i][j] = ((DATA_TYPE) (i-1)*(j+2) + 2) / NX;
			hz[i][j] = ((DATA_TYPE) (i-9)*(j+4) + 3) / NX;
		}
	}
}


void runFdtd(int tmax, int nx, int ny, DATA_TYPE POLYBENCH_1D(_fict_, TMAX, TMAX), DATA_TYPE POLYBENCH_2D(ex,NX,NY,nx,ny), 
	DATA_TYPE POLYBENCH_2D(ey,NX,NY,nx,ny), DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny))
{
	int t, i, j;
	
	for (t=0; t < _PB_TMAX; t++)  
	{
		for (j=0; j < _PB_NY; j++)
		{
			ey[0][j] = _fict_[t];
		}
	
		for (i = 1; i < _PB_NX; i++)
		{
       		for (j = 0; j < _PB_NY; j++)
			{
       			ey[i][j] = ey[i][j] - 0.5*(hz[i][j] - hz[(i-1)][j]);
        		}
		}

		for (i = 0; i < _PB_NX; i++)
		{
       		for (j = 1; j < _PB_NY; j++)
			{
				ex[i][j] = ex[i][j] - 0.5*(hz[i][j] - hz[i][(j-1)]);
			}
		}

		for (i = 0; i < _PB_NX-1; i++)
		{
			for (j = 0; j < _PB_NY-1; j++)
			{
				hz[i][j] = hz[i][j] - 0.7*(ex[i][(j+1)] - ex[i][j] + ey[(i+1)][j] - ey[i][j]);
			}
		}
	}
}


void compareResults(int nx, int ny, DATA_TYPE POLYBENCH_2D(hz1,NX,NY,nx,ny), DATA_TYPE POLYBENCH_2D(hz2,NX,NY,nx,ny))
{
	int i, j, fail;
	fail = 0;
	
	for (i=0; i < nx; i++) 
	{
		for (j=0; j < ny; j++) 
		{
			if (percentDiff(hz1[i][j], hz2[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	//printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}



__global__ void fdtd_step1_kernel(int nx, int ny, DATA_TYPE* _fict_, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < _PB_NX) && (j < _PB_NY))
	{
		if (i == 0) 
		{
			ey[i * NY + j] = _fict_[t];
		}
		else
		{ 
			ey[i * NY + j] = ey[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[(i-1) * NY + j]);
		}
	}
}



__global__ void fdtd_step2_kernel(int nx, int ny, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < _PB_NX) && (j < _PB_NY) && (j > 0))
	{
		ex[i * NY + j] = ex[i * NY + j] - 0.5f*(hz[i * NY + j] - hz[i * NY + (j-1)]);
	}
}


__global__ void fdtd_step3_kernel(int nx, int ny, DATA_TYPE *ex, DATA_TYPE *ey, DATA_TYPE *hz, int t)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ((i < (_PB_NX-1)) && (j < (_PB_NY-1)))
	{	
		hz[i * NY + j] = hz[i * NY + j] - 0.7f*(ex[i * NY + (j+1)] - ex[i * NY + j] + ey[(i + 1) * NY + j] - ey[i * NY + j]);
	}
}


void fdtdCuda(int tmax, int nx, int ny, DATA_TYPE POLYBENCH_1D(_fict_, TMAX, TMAX), DATA_TYPE POLYBENCH_2D(ex,NX,NY,nx,ny), 
	DATA_TYPE POLYBENCH_2D(ey,NX,NY,nx,ny), DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny), DATA_TYPE POLYBENCH_2D(hz_outputFromGpu,NX,NY,nx,ny))
{
	DATA_TYPE *_fict_gpu;
	DATA_TYPE *ex_gpu;
	DATA_TYPE *ey_gpu;
	DATA_TYPE *hz_gpu;

	start = clock();
	hipMalloc((void **)&_fict_gpu, sizeof(DATA_TYPE) * TMAX);
	hipMalloc((void **)&ex_gpu, sizeof(DATA_TYPE) * NX * NY);
	hipMalloc((void **)&ey_gpu, sizeof(DATA_TYPE) * NX * NY);
	hipMalloc((void **)&hz_gpu, sizeof(DATA_TYPE) * NX * NY);
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();

	hipMemcpy(_fict_gpu, _fict_, sizeof(DATA_TYPE) * TMAX, hipMemcpyHostToDevice);
	hipMemcpy(ex_gpu, ex, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
	hipMemcpy(ey_gpu, ey, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
	hipMemcpy(hz_gpu, hz, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid( (size_t)ceil(((float)NY) / ((float)block.x)), (size_t)ceil(((float)NX) / ((float)block.y)));
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	for(int t = 0; t < _PB_TMAX; t++)
	{
		fdtd_step1_kernel<<<grid,block>>>(nx, ny, _fict_gpu, ex_gpu, ey_gpu, hz_gpu, t);
		hipDeviceSynchronize();
		fdtd_step2_kernel<<<grid,block>>>(nx, ny, ex_gpu, ey_gpu, hz_gpu, t);
		hipDeviceSynchronize();
		fdtd_step3_kernel<<<grid,block>>>(nx, ny, ex_gpu, ey_gpu, hz_gpu, t);
		hipDeviceSynchronize();
	}
	
	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();

	hipMemcpy(hz_outputFromGpu, hz_gpu, sizeof(DATA_TYPE) * NX * NY, hipMemcpyDeviceToHost);
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();	
		
	hipFree(_fict_gpu);
	hipFree(ex_gpu);
	hipFree(ey_gpu);
	hipFree(hz_gpu);
	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nx,
		 int ny,
		 DATA_TYPE POLYBENCH_2D(hz,NX,NY,nx,ny))
{
  int i, j;

  for (i = 0; i < nx; i++)
    for (j = 0; j < ny; j++) {
         fprintf(stderr, DATA_PRINTF_MODIFIER, hz[i][j]);
      if ((i * nx + j) % 20 == 0) fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}


void *poly_main(void *x)
{
	start = clock();

	int tmax = TMAX;
	int nx = NX;
	int ny = NY;

	POLYBENCH_1D_ARRAY_DECL(_fict_,DATA_TYPE,TMAX,TMAX);
	POLYBENCH_2D_ARRAY_DECL(ex,DATA_TYPE,NX,NY,nx,ny);
	POLYBENCH_2D_ARRAY_DECL(ey,DATA_TYPE,NX,NY,nx,ny);
	POLYBENCH_2D_ARRAY_DECL(hz,DATA_TYPE,NX,NY,nx,ny);
	POLYBENCH_2D_ARRAY_DECL(hz_outputFromGpu,DATA_TYPE,NX,NY,nx,ny);

	init_arrays(tmax, nx, ny, POLYBENCH_ARRAY(_fict_), POLYBENCH_ARRAY(ex), POLYBENCH_ARRAY(ey), POLYBENCH_ARRAY(hz));

	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
    
	GPU_argv_init();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;

	fdtdCuda(tmax, nx, ny, POLYBENCH_ARRAY(_fict_), POLYBENCH_ARRAY(ex), POLYBENCH_ARRAY(ey), POLYBENCH_ARRAY(hz), POLYBENCH_ARRAY(hz_outputFromGpu));

	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		runFdtd(tmax, nx, ny, POLYBENCH_ARRAY(_fict_), POLYBENCH_ARRAY(ex), POLYBENCH_ARRAY(ey), POLYBENCH_ARRAY(hz));

		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;
		
		compareResults(nx, ny, POLYBENCH_ARRAY(hz), POLYBENCH_ARRAY(hz_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(nx, ny, POLYBENCH_ARRAY(hz_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();

	POLYBENCH_FREE_ARRAY(_fict_);
	POLYBENCH_FREE_ARRAY(ex);
	POLYBENCH_FREE_ARRAY(ey);
	POLYBENCH_FREE_ARRAY(hz);
	POLYBENCH_FREE_ARRAY(hz_outputFromGpu);

	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);

	return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}
#include <polybench.c>