#include "hip/hip_runtime.h"
/**
 * jacobi1D.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "jacobi1D.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void init_array(int n, DATA_TYPE POLYBENCH_1D(A,N,n), DATA_TYPE POLYBENCH_1D(B,N,n))
{
	int i;

	for (i = 0; i < n; i++)
    	{
		A[i] = ((DATA_TYPE) 4 * i + 10) / N;
		B[i] = ((DATA_TYPE) 7 * i + 11) / N;
    	}
}


void runJacobi1DCpu(int tsteps, int n, DATA_TYPE POLYBENCH_1D(A,N,n), DATA_TYPE POLYBENCH_1D(B,N,n))
{
	for (int t = 0; t < _PB_TSTEPS; t++)
    {
		for (int i = 2; i < _PB_N - 1; i++)
		{
			B[i] = 0.33333 * (A[i-1] + A[i] + A[i + 1]);
		}
		
		for (int j = 2; j < _PB_N - 1; j++)
		{
			A[j] = B[j];
		}
    }
}


__global__ void runJacobiCUDA_kernel1(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((i > 1) && (i < (_PB_N-1)))
	{
		B[i] = 0.33333f * (A[i-1] + A[i] + A[i + 1]);
	}
}


__global__ void runJacobiCUDA_kernel2(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((j > 1) && (j < (_PB_N-1)))
	{
		A[j] = B[j];
	}
}


void compareResults(int n, DATA_TYPE POLYBENCH_1D(a,N,n), DATA_TYPE POLYBENCH_1D(a_outputFromGpu,N,n), DATA_TYPE POLYBENCH_1D(b,N,n), DATA_TYPE POLYBENCH_1D(b_outputFromGpu,N,n))
{
	int i, fail;
	fail = 0;   

	// Compare a and c
	for (i=0; i < n; i++) 
	{
		if (percentDiff(a[i], a_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD) 
		{
			fail++;
		}
	}

	for (i=0; i < n; i++) 
	{
		if (percentDiff(b[i], b_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD) 
		{	
			fail++;
		}
	}

	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void runJacobi1DCUDA(int tsteps, int n, DATA_TYPE POLYBENCH_1D(A,N,n), DATA_TYPE POLYBENCH_1D(B,N,n), DATA_TYPE POLYBENCH_1D(A_outputFromGpu,N,n), 
			DATA_TYPE POLYBENCH_1D(B_outputFromGpu,N,n))
{
	DATA_TYPE* Agpu;
	DATA_TYPE* Bgpu;
	start = clock();
	hipMalloc(&Agpu, N * sizeof(DATA_TYPE));
	hipMalloc(&Bgpu, N * sizeof(DATA_TYPE));
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(Agpu, A, N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
	hipMemcpy(Bgpu, B, N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)N) / ((float)block.x) ), 1);
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	for (int t = 0; t < _PB_TSTEPS ; t++)
	{
		runJacobiCUDA_kernel1 <<< grid, block >>> (n, Agpu, Bgpu);
		hipDeviceSynchronize();
		runJacobiCUDA_kernel2 <<< grid, block>>> (n, Agpu, Bgpu);
		hipDeviceSynchronize();
	}

	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	
	hipMemcpy(A_outputFromGpu, Agpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);
	hipMemcpy(B_outputFromGpu, Bgpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();
	hipFree(Agpu);
	hipFree(Bgpu);
	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(A,N,n))

{
  int i;

  for (i = 0; i < n; i++)
    {
      fprintf(stderr, DATA_PRINTF_MODIFIER, A[i]);
      if (i % 20 == 0) fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}


void *poly_main(void *x)
{
	start = clock();
	/* Retrieve problem size. */
	int n = N;
	int tsteps = TSTEPS;

	POLYBENCH_1D_ARRAY_DECL(a,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(b,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(a_outputFromGpu,DATA_TYPE,N,n);
	POLYBENCH_1D_ARRAY_DECL(b_outputFromGpu,DATA_TYPE,N,n);

	init_array(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b));

	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;

	runJacobi1DCUDA(tsteps, n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b), POLYBENCH_ARRAY(a_outputFromGpu), POLYBENCH_ARRAY(b_outputFromGpu));

	#ifdef RUN_ON_CPU
	
		/* Start timer. */
	  	polybench_start_instruments;
	
		runJacobi1DCpu(tsteps, n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b));
	
		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;

		compareResults(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(a_outputFromGpu), POLYBENCH_ARRAY(b), POLYBENCH_ARRAY(b_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(a_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();
	POLYBENCH_FREE_ARRAY(a);
	POLYBENCH_FREE_ARRAY(a_outputFromGpu);
	POLYBENCH_FREE_ARRAY(b);
	POLYBENCH_FREE_ARRAY(b_outputFromGpu);
	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);


	return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}

#include <polybench.c>