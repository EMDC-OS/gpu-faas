#include "hip/hip_runtime.h"
/**
 * jacobi2D.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>
#include <pthread.h>

#define POLYBENCH_TIME 1

#include "jacobi2D.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

//#define RUN_ON_CPU

// hdi=host_data_init, kdi=kernel-related_data_init gset=gpu_set, 
clock_t start, end;
double t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear;


void init_array(int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n))
{
	int i, j;

	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{
			A[i][j] = ((DATA_TYPE) i*(j+2) + 10) / N;
			B[i][j] = ((DATA_TYPE) (i-4)*(j-1) + 11) / N;
		}
	}
}


void runJacobi2DCpu(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n))
{
	for (int t = 0; t < _PB_TSTEPS; t++)
	{
    		for (int i = 1; i < _PB_N - 1; i++)
		{
			for (int j = 1; j < _PB_N - 1; j++)
			{
	  			B[i][j] = 0.2f * (A[i][j] + A[i][(j-1)] + A[i][(1+j)] + A[(1+i)][j] + A[(i-1)][j]);
			}
		}
		
    		for (int i = 1; i < _PB_N-1; i++)
		{
			for (int j = 1; j < _PB_N-1; j++)
			{
	  			A[i][j] = B[i][j];
			}
		}
	}
}


__global__ void runJacobiCUDA_kernel1(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if ((i >= 1) && (i < (_PB_N-1)) && (j >= 1) && (j < (_PB_N-1)))
	{
		B[i*N + j] = 0.2f * (A[i*N + j] + A[i*N + (j-1)] + A[i*N + (1 + j)] + A[(1 + i)*N + j] + A[(i-1)*N + j]);	
	}
}


__global__ void runJacobiCUDA_kernel2(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((i >= 1) && (i < (_PB_N-1)) && (j >= 1) && (j < (_PB_N-1)))
	{
		A[i*N + j] = B[i*N + j];
	}
}


void compareResults(int n, DATA_TYPE POLYBENCH_2D(a,N,N,n,n), DATA_TYPE POLYBENCH_2D(a_outputFromGpu,N,N,n,n), DATA_TYPE POLYBENCH_2D(b,N,N,n,n), DATA_TYPE POLYBENCH_2D(b_outputFromGpu,N,N,n,n))
{
	int i, j, fail;
	fail = 0;   

	// Compare output from CPU and GPU
	for (i=0; i<n; i++) 
	{
		for (j=0; j<n; j++) 
		{
			if (percentDiff(a[i][j], a_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
        }
	}
  
	for (i=0; i<n; i++) 
	{
       	for (j=0; j<n; j++) 
		{
        		if (percentDiff(b[i][j], b_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
        			fail++;
        		}
       	}
	}

	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void runJacobi2DCUDA(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n), DATA_TYPE POLYBENCH_2D(A_outputFromGpu,N,N,n,n), DATA_TYPE POLYBENCH_2D(B_outputFromGpu,N,N,n,n))
{
	DATA_TYPE* Agpu;
	DATA_TYPE* Bgpu;
	start = clock();
	hipMalloc(&Agpu, N * N * sizeof(DATA_TYPE));
	hipMalloc(&Bgpu, N * N * sizeof(DATA_TYPE));
	end = clock();
	t_malloc = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	hipMemcpy(Agpu, A, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
	hipMemcpy(Bgpu, B, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
	end = clock();
	t_write = (double)(end-start) / CLOCKS_PER_SEC;
		
	start = clock();

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)N) / ((float)block.x) ), (unsigned int)ceil( ((float)N) / ((float)block.y) ));
	end = clock();
	t_kdi = (double)(end-start) / CLOCKS_PER_SEC;

	/* Start timer. */
  	//polybench_start_instruments;
	start = clock();

	for (int t = 0; t < _PB_TSTEPS; t++)
	{
		runJacobiCUDA_kernel1<<<grid,block>>>(n, Agpu, Bgpu);
		hipDeviceSynchronize();
		runJacobiCUDA_kernel2<<<grid,block>>>(n, Agpu, Bgpu);
		hipDeviceSynchronize();
	}

	/* Stop and print timer. */
	//printf("GPU Time in seconds:\n");
  	//polybench_stop_instruments;
 	//polybench_print_instruments;
	end = clock();
	t_kernel = (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();
	
	hipMemcpy(A_outputFromGpu, Agpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
	hipMemcpy(B_outputFromGpu, Bgpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
	end = clock();
	t_read = (double)(end-start) / CLOCKS_PER_SEC;
	
	start = clock();

	hipFree(Agpu);
	hipFree(Bgpu);
	end = clock();
	t_clear = (double)(end-start) / CLOCKS_PER_SEC;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n))

{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
      fprintf(stderr, DATA_PRINTF_MODIFIER, A[i][j]);
      if ((i * n + j) % 20 == 0) fprintf(stderr, "\n");
    }
  fprintf(stderr, "\n");
}


void *poly_main(void *x)
{
	start = clock();
	/* Retrieve problem size. */
	int n = N;
	int tsteps = TSTEPS;

	POLYBENCH_2D_ARRAY_DECL(a,DATA_TYPE,N,N,n,n);
	POLYBENCH_2D_ARRAY_DECL(b,DATA_TYPE,N,N,n,n);
	POLYBENCH_2D_ARRAY_DECL(a_outputFromGpu,DATA_TYPE,N,N,n,n);
	POLYBENCH_2D_ARRAY_DECL(b_outputFromGpu,DATA_TYPE,N,N,n,n);

	init_array(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b));
	end = clock();
	t_hdi += (double)(end-start) / CLOCKS_PER_SEC;

	start = clock();

	end = clock();
	t_gset += (double)(end-start) / CLOCKS_PER_SEC;
	runJacobi2DCUDA(tsteps, n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b), POLYBENCH_ARRAY(a_outputFromGpu), POLYBENCH_ARRAY(b_outputFromGpu));

	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		runJacobi2DCpu(tsteps, n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b));
	
		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	  	polybench_print_instruments;
	
		compareResults(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(a_outputFromGpu), POLYBENCH_ARRAY(b), POLYBENCH_ARRAY(b_outputFromGpu));

	#else //prevent dead code elimination

		//polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(a_outputFromGpu)));

	#endif //RUN_ON_CPU

	start = clock();
	POLYBENCH_FREE_ARRAY(a);
	POLYBENCH_FREE_ARRAY(a_outputFromGpu);
	POLYBENCH_FREE_ARRAY(b);
	POLYBENCH_FREE_ARRAY(b_outputFromGpu);
	end = clock();
	t_clear += (double)(end-start) / CLOCKS_PER_SEC;

	printf("{\"hdi\": %lf, \"gset\": %lf, \"malloc\": %lf, \"write\": %lf, \"kdi\": %lf, \"kernel\": %lf, \"read\": %lf, \"clear\": %lf}",
		t_hdi, t_gset, t_malloc, t_write, t_kdi, t_kernel, t_read, t_clear);


	return 0;
}

int main()
{
	int tid, status;
	int *init_mem = 0;
	pthread_t thread_id;

	//start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&init_mem, 4096 * sizeof(int));
	//end = clock();
	//printf("Master overhead: %lf\n",(double)(end-start) / CLOCKS_PER_SEC);

	tid = pthread_create(&thread_id, NULL, poly_main, NULL);
	pthread_join(thread_id, (void**)&status);

	hipFree(init_mem);

	return 0;
}

#include <polybench.c>