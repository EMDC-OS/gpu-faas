
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <time.h>
#include <dlfcn.h>

int main(void)
{
	int *temp = 0;
	clock_t start, end;

	void *handle_2mm;
	void *handle_3mm;
	void *handle_atax;
	void *handle_bicg;
	void *handle_doitgen;
	void *handle_gemm;
	void *handle_gemver;
	void *handle_gesummv;
	void *handle_mvt;
	void *handle_syr2k;
	void *handle_syrk;
	void *handle_adi;
	void *handle_conv2d;
	void *handle_conv3d;
	void *handle_fdtd;
	void *handle_jacobi1d;
	void *handle_jacobi2d;

	int (*test_2mm)(int);
	int (*test_3mm)(int);
	int (*test_atax)(int);
	int (*test_bicg)(int);
	int (*test_doitgen)(int);
	int (*test_gemm)(int);
	int (*test_gemver)(int);
	int (*test_gesummv)(int);
	int (*test_mvt)(int);
	int (*test_syr2k)(int);
	int (*test_syrk)(int);
	int (*test_adi)(int);
	int (*test_conv2d)(int);
	int (*test_conv3d)(int);
	int (*test_fdtd)(int);
	int (*test_jacobi1d)(int);
	int (*test_jacobi2d)(int);

	start = clock();
	hipSetDevice(0);
	hipMalloc((void**)&temp, sizeof(int));
	end = clock();

	printf("Wait 1 seconds\n");
	usleep(1000000);



	/* 2mm */
	handle_2mm = dlopen("./2mm.so", RTLD_LAZY);
	if (!handle_2mm) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_2mm) = dlsym(handle_2mm, "_Z8main_2mmi");

	handle_2mm = dlopen("./2mm.so", RTLD_LAZY);
	if (!handle_2mm) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_2mm)(5);
	dlclose(handle_2mm);


	/* 3mm */
	handle_3mm = dlopen("./3mm.so", RTLD_LAZY);
	if (!handle_3mm) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_3mm) = dlsym(handle_3mm, "_Z8main_3mmi");

	handle_3mm = dlopen("./3mm.so", RTLD_LAZY);
	if (!handle_3mm) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_3mm)(5);
	dlclose(handle_3mm);


	/* atax */
	handle_atax = dlopen("./atax.so", RTLD_LAZY);
	if (!handle_atax) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_atax) = dlsym(handle_atax, "_Z9main_ataxi");

	handle_atax = dlopen("./atax.so", RTLD_LAZY);
	if (!handle_atax) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_atax)(5);
	dlclose(handle_atax);


	/* bicg */
	handle_bicg = dlopen("./bicg.so", RTLD_LAZY);
	if (!handle_bicg) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_bicg) = dlsym(handle_bicg, "_Z9main_bicgi");

	handle_bicg = dlopen("./bicg.so", RTLD_LAZY);
	if (!handle_bicg) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_bicg)(5);
	dlclose(handle_bicg);


	/* doitgen */
	handle_doitgen = dlopen("./doitgen.so", RTLD_LAZY);
	if (!handle_doitgen) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_doitgen) = dlsym(handle_doitgen, "_Z12main_doitgeni");

	handle_doitgen = dlopen("./doitgen.so", RTLD_LAZY);
	if (!handle_doitgen) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_doitgen)(5);
	dlclose(handle_doitgen);


	/* gemm */
	handle_gemm = dlopen("./gemm.so", RTLD_LAZY);
	if (!handle_gemm) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_gemm) = dlsym(handle_gemm, "_Z9main_gemmi");

	handle_gemm = dlopen("./gemm.so", RTLD_LAZY);
	if (!handle_gemm) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_gemm)(5);
	dlclose(handle_gemm);


	/* gemver */
	handle_gemver = dlopen("./gemver.so", RTLD_LAZY);
	if (!handle_gemver) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_gemver) = dlsym(handle_gemver, "_Z11main_gemveri");

	handle_gemver = dlopen("./gemver.so", RTLD_LAZY);
	if (!handle_gemver) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_gemver)(5);
	dlclose(handle_gemver);


	/* gesummv */
	handle_gesummv = dlopen("./gesummv.so", RTLD_LAZY);
	if (!handle_gesummv) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_gesummv) = dlsym(handle_gesummv, "_Z12main_gesummvi");

	handle_gesummv = dlopen("./gesummv.so", RTLD_LAZY);
	if (!handle_gesummv) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_gesummv)(5);
	dlclose(handle_gesummv);


	/* mvt */
	handle_mvt = dlopen("./mvt.so", RTLD_LAZY);
	if (!handle_mvt) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_mvt) = dlsym(handle_mvt, "_Z8main_mvti");

	handle_mvt = dlopen("./mvt.so", RTLD_LAZY);
	if (!handle_mvt) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_mvt)(5);
	dlclose(handle_mvt);


	/* syr2k */
	handle_syr2k = dlopen("./syr2k.so", RTLD_LAZY);
	if (!handle_syr2k) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_syr2k) = dlsym(handle_syr2k, "_Z10main_syr2ki");

	handle_syr2k = dlopen("./syr2k.so", RTLD_LAZY);
	if (!handle_syr2k) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_syr2k)(5);
	dlclose(handle_syr2k);


	/* syrk */
	handle_syrk = dlopen("./syrk.so", RTLD_LAZY);
	if (!handle_syrk) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_syrk) = dlsym(handle_syrk, "_Z9main_syrki");

	handle_syrk = dlopen("./syrk.so", RTLD_LAZY);
	if (!handle_syrk) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_syrk)(5);
	dlclose(handle_syrk);


	/* adi */
	handle_adi = dlopen("./adi.so", RTLD_LAZY);
	if (!handle_adi) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_adi) = dlsym(handle_adi, "_Z8main_adii");

	handle_adi = dlopen("./adi.so", RTLD_LAZY);
	if (!handle_adi) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_adi)(5);
	dlclose(handle_adi);


	/* conv2d */
	handle_conv2d = dlopen("./2DConvolution.so", RTLD_LAZY);
	if (!handle_conv2d) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_conv2d) = dlsym(handle_conv2d, "_Z11main_conv2di");

	handle_conv2d = dlopen("./2DConvolution.so", RTLD_LAZY);
	if (!handle_conv2d) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_conv2d)(5);
	dlclose(handle_conv2d);


	/* conv3d */
	handle_conv3d = dlopen("./3DConvolution.so", RTLD_LAZY);
	if (!handle_conv3d) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_conv3d) = dlsym(handle_conv3d, "_Z11main_conv3di");

	handle_conv3d = dlopen("./3DConvolution.so", RTLD_LAZY);
	if (!handle_conv3d) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	(*test_conv3d)(5);
	dlclose(handle_conv3d);


	/* fdtd */
	handle_fdtd = dlopen("./fdtd2d.so", RTLD_LAZY);
	if (!handle_fdtd) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_fdtd) = dlsym(handle_fdtd, "_Z9main_fdtdi");

	handle_fdtd = dlopen("./fdtd2d.so", RTLD_LAZY);
	if (!handle_fdtd) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	printf("%d main time = %lf\n", (*test_fdtd)(5), (float)(end-start)/CLOCKS_PER_SEC);
	dlclose(handle_fdtd);


	/* jacobi1d */
	handle_jacobi1d = dlopen("./jacobi1D.so", RTLD_LAZY);
	if (!handle_jacobi1d) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_jacobi1d) = dlsym(handle_jacobi1d, "_Z13main_jacobi1di");

	handle_jacobi1d = dlopen("./jacobi1D.so", RTLD_LAZY);
	if (!handle_jacobi1d) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	printf("%d main time = %lf\n", (*test_jacobi1d)(5), (float)(end-start)/CLOCKS_PER_SEC);
	dlclose(handle_jacobi1d);


	/* jacobi2d */
	handle_jacobi2d = dlopen("./jacobi2D.so", RTLD_LAZY);
	if (!handle_jacobi2d) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	dlerror();

	*(void **)(&test_jacobi2d) = dlsym(handle_jacobi2d, "_Z13main_jacobi2di");

	handle_jacobi2d = dlopen("./jacobi2D.so", RTLD_LAZY);
	if (!handle_jacobi2d) {
		fprintf(stderr, "%s\n", dlerror());
		exit(EXIT_FAILURE);
	}

	printf("%d main time = %lf\n", (*test_jacobi2d)(5), (float)(end-start)/CLOCKS_PER_SEC);
	dlclose(handle_jacobi2d);


	return 0;
}

