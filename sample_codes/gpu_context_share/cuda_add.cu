#include "hip/hip_runtime.h"


#include <stdio.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <time.h>

#define arraySize 1000

struct HUGE {
	int x[500000];
};

__global__ void addKernel( int *c, const int *a, const int *b )
{
    int i = threadIdx.x;

	if( i < arraySize )
		c[i] = a[i] + b[i];
}

int main()
{
    int a[arraySize];
    int b[arraySize];
    int c[arraySize];
    int x;

    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    int *dmp = 0;

    pid_t pid;
    clock_t start, end;

    // fill the arrays 'a' and 'b' on the CPU
    for( int i = 0 ; i < arraySize ; i++ ) {
	a[i] = i;
 	b[i] = i;
    }
    scanf("%d",&x);
    hipSetDevice(0);
    hipMalloc((void**)&dmp, sizeof(int));
    pid = fork();
    printf("%d\n",(int)pid);
    if (pid == 0){
    	// Add vectors in parallel.
    	// Allocate GPU buffers for three vectors (two input, one output)
	scanf("%d",&x);

	start = clock();
    	hipMalloc((void**)&dev_c, arraySize * sizeof(struct HUGE));
    	hipMalloc((void**)&dev_a, arraySize * sizeof(struct HUGE));
    	hipMalloc((void**)&dev_b, arraySize * sizeof(struct HUGE));
	end = clock();
	printf("pid: %d, child time %lf\n",(int)getpid(), (double)(end-start)/CLOCKS_PER_SEC);


    	// copy the arrays 'a' and 'b' to the GPU
    	hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

	//scanf("%d",&x);

    	addKernel<<<1, arraySize>>>(dev_c, dev_a, dev_b);
    	hipDeviceSynchronize();

	// copy the array 'c' back from the GPU to the CPU
	hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

        printf( "Child %d + %d = %d\n", a[126], b[126], c[126] );
    }
    else{
	start = clock();
        hipMalloc((void**)&dev_c, arraySize * sizeof(struct HUGE));
        hipMalloc((void**)&dev_a, arraySize * sizeof(struct HUGE));
        hipMalloc((void**)&dev_b, arraySize * sizeof(struct HUGE));
	end = clock();
	printf("pid: %d, parent time %lf\n",(int)getpid(), (double)(end-start)/CLOCKS_PER_SEC);

        // copy the arrays 'a' and 'b' to the GPU
        hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

        addKernel<<<1, arraySize>>>(dev_c, dev_a, dev_b);
        hipDeviceSynchronize();

        // copy the array 'c' back from the GPU to the CPU
        hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

	printf( "Parent %d + %d = %d\n", a[126], b[126], c[126] );

	wait(NULL);
	// display the results
    }

    //scanf("%d", &x);

    // free the memory allocated on the GPU
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dmp);
    
    return 0;
}
